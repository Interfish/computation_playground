#include "hip/hip_runtime.h"
namespace computation_playground {

void transpose2d_naive(float *tensor, float* out, int m, int n) {
  dim3 block(1);
  transpose2d_naive<<<1, block>>>(tensor, out, m, n);
}

__global__ void transpose2d_naive(float *tensor, float* out, int m, int n) { }

}