#include <cmath>

#include <hip/hip_runtime.h>

namespace computation_playground {

__global__ void transpose2d_naive_kernel(float* in, float* out, int m, int n) {
  int in_row_offet = blockIdx.x * blockDim.x + threadIdx.x;
  if(in_row_offet < m) {
    int in_global_offset = blockIdx.y * m + in_row_offet;
    int out_global_offset = in_row_offet * n + blockIdx.y;
    *(out + out_global_offset) = *(in + in_global_offset);
  }
}

void transpose2d_naive(float* in, float* out, int m, int n, hipStream_t stream) {
  int threads_per_block = 32;
  int blocks_per_row = std::ceil(m / float(threads_per_block));
  dim3 grid(blocks_per_row, n);
  transpose2d_naive_kernel<<<grid, threads_per_block, 0, stream>>>(in, out, m, n);
}

__global__ void transpose2d_tile_kernel(float* in, float* out, int m, int n,
                                        int m_direction_iteration, int n_direction_iteration) {
  for(int j = 0; j < n_direction_iteration; j++) {
    for(int i = 0; i < m_direction_iteration; i++) {
      int x = blockIdx.x * blockDim.x * m_direction_iteration + i * blockDim.x + threadIdx.x;
      int y = blockIdx.y * blockDim.y * n_direction_iteration + j * blockDim.y + threadIdx.y;
      if(x < m && y < n) {
        out[x * n + y] = in[y * m + x];
      }
    }
  }
}

void transpose2d_tile(float* in, float* out, int m, int n, int tile_m_dim, int tile_n_dim,
                      int m_direction_iteration, int n_direction_iteration, hipStream_t stream) {
  dim3 grid(std::ceil(m / (tile_m_dim * m_direction_iteration)), std::ceil(n / (tile_n_dim * n_direction_iteration)));
  dim3 block(tile_m_dim, tile_n_dim);
  transpose2d_tile_kernel<<<grid, block, 0, stream>>>(in, out, m, n, m_direction_iteration, n_direction_iteration);
}

} // namespace computation_playground