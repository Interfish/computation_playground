#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <ctime>
#include "hip/hip_runtime.h"

#include "common.cuh"
#include "gemmk.cuh"

int main(int argc, char *argv[]) {
    int m, k, n, kernelSize;
    m = atoi(argv[1]);
    k = atoi(argv[2]);
    n = atoi(argv[3]);
    kernelSize = atoi(argv[4]);

    float* a = new float[m * k];
    float* b = new float[k * n];
    float* c = new float[m * n];
    float *ad, *bd, *cd;

    for (int i=0; i < m*k; i++) {
        a[i] = 1.0;
    }
    for (int i=0;i<k*n;i++) {
        b[i] = 1.0;
    }
    gpuErrchk(hipMalloc((void**)&ad, m * k * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&bd, k * n * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&cd, m * n * sizeof(float)));

    gpuErrchk(hipMemcpy(ad, a, m * k * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(bd, b, n * k * sizeof(float), hipMemcpyHostToDevice));


    dim3 block(kernelSize, kernelSize);
    dim3 grid(ceil(float(n) / kernelSize), ceil(float(m) / kernelSize));


    hipEvent_t start, stop;
    float elapsedTime;
    // gemmFast1
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    gemmFast1<<<grid, block, 2 * kernelSize * kernelSize * sizeof(float)>>>(ad, bd, cd, m, k, n);
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << std::fixed << std::setprecision(2) << "gemmFast kernel time used: "
              << elapsedTime << "ms" << std::endl;
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++) {
        //printf("%d, %f\n", i, c[i]);
        // assert(c[i] == k);
    }


    // gemmVanilla
    gemmVanilla<<<grid, block>>>(ad, bd, cd, m, k, n);
    // std::cout << std::fixed << std::setprecision(2) << "gemmVanilla kernel time used: "
    //           << double(c_end - c_start) / CLOCKS_PER_SEC << " s" << std::endl;
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++)
        assert(c[i] == k);

    // matrixMul_noBankConflict
    // matrixMul_noBankConflict<<<grid, block>>>(cd, ad, bd, k, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++)
        assert(c[i] == k);

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    delete []a;
    delete []b;
    delete []c;
}