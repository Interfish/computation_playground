#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"

#include "common.cuh"

__global__ void gemm(float* a, float* b, float* c, int m, int k, int n) {
    // a is a m x k matrix
    // b is a k x n matrix
    extern __shared__ float Tile[];
    float *aTile = Tile;
    float *bTile = Tile + blockDim.x * blockDim.y;

    // int tid = blockDim.x * threadIdx.y + threadIdx.x;
    // printf("%d %d %d\n", blockIdx.y, blockIdx.x, tid);

    int globalXA, globalYA, globalXB, globalYB, globalXC, globalYC;
    float *aPoint, *bPoint, accu = 0.0;
    for (int tileStart = 0; tileStart < k; tileStart += blockDim.x) {
        globalXA = tileStart + threadIdx.x;
        globalYA = blockIdx.y * blockDim.y + threadIdx.y;
        globalXB = blockIdx.x * blockDim.x + threadIdx.x;
        globalYB = tileStart + threadIdx.y;
        aPoint = aTile + threadIdx.y * blockDim.x + threadIdx.x;
        bPoint = bTile + threadIdx.y * blockDim.x + threadIdx.x;
        // printf("%d\n", tileStart);

        // printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);

        // printf("%d %d %d %d \n", globalXA, globalYA, globalXB, globalYB);

        if (globalXA < k && globalYA < m) {
            *(aPoint) = *(a + globalYA * k + globalXA);
        } else {
            *(aPoint) = 0.0;
            // printf("%d %d %d %d \n", globalXA, globalYA, globalXB, globalYB);
        }
        if (globalXB < n && globalYB < k) {
            *(bPoint) = *(b + globalYB * n + globalXB);
        } else {
            *(bPoint) = 0.0;
        }
        __syncthreads();
        //printf("%d %d %d %d %f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, *(aPoint));

        for (int i = 0; i < blockDim.x; i++) {
            accu += aTile[threadIdx.y * blockDim.x + i] * bTile[i * blockDim.x + threadIdx.x];
        }
        __syncthreads();
    }
    globalXC = blockIdx.x * blockDim.x + threadIdx.x;
    globalYC = blockIdx.y * blockDim.y + threadIdx.y;
    if(globalXC < n && globalYC < m) {
        c[globalYC * n + globalXC] = accu;
    }
}

int main(int argc, char *argv[]) {
    int m, k, n, kernelSize;
    m = atoi(argv[1]);
    k = atoi(argv[2]);
    n = atoi(argv[3]);
    kernelSize = atoi(argv[4]);

    float* a = new float[m * k];
    float* b = new float[k * n];
    float* c = new float[m * n];
    float *ad, *bd, *cd;

    for (int i=0; i < m*k; i++) {
        a[i] = 1.0;
    }
    for (int i=0;i<k*n;i++) {
        b[i] = 1.0;
    }
    gpuErrchk(hipMalloc((void**)&ad, m * k * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&bd, k * n * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&cd, m * n * sizeof(float)));

    gpuErrchk(hipMemcpy(ad, a, m * k * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(bd, b, n * k * sizeof(float), hipMemcpyHostToDevice));

    dim3 block(kernelSize, kernelSize);
    dim3 grid(ceil(float(m) / kernelSize), ceil(float(n) / kernelSize));

    // std::cout << ceil(float(m) / kernelSize) << ' ' << ceil(float(n) / kernelSize) << std::endl;

    gemm<<<grid, block, 2 * kernelSize * kernelSize * sizeof(float)>>>(ad, bd, cd, m, k, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++) {
        assert(c[i] == k);
    }

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    delete []a;
    delete []b;
    delete []c;
}