#include "hip/hip_runtime.h"
#include <random>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"

#include "common.cuh"
#include "gemmk.cuh"

int main(int argc, char *argv[]) {
    int m, k, n, kernelSize;
    m = atoi(argv[1]);
    k = atoi(argv[2]);
    n = atoi(argv[3]);
    kernelSize = atoi(argv[4]);

    float* a = new float[m * k];
    float* b = new float[k * n];
    float* c = new float[m * n];
    float *ad, *bd, *cd;

    for (int i=0; i < m*k; i++) {
        a[i] = 1.0;
    }
    for (int i=0;i<k*n;i++) {
        b[i] = 1.0;
    }
    gpuErrchk(hipMalloc((void**)&ad, m * k * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&bd, k * n * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&cd, m * n * sizeof(float)));

    gpuErrchk(hipMemcpy(ad, a, m * k * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(bd, b, n * k * sizeof(float), hipMemcpyHostToDevice));


    dim3 block(kernelSize, kernelSize);
    dim3 grid(ceil(float(n) / kernelSize), ceil(float(m) / kernelSize));


    // gemmFast1
    gemmFast1<<<grid, block, 2 * kernelSize * kernelSize * sizeof(float)>>>(ad, bd, cd, m, k, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++) {
        //printf("%d, %f\n", i, c[i]);
        // assert(c[i] == k);
    }


    // gemmVanilla
    gemmVanilla<<<grid, block>>>(ad, bd, cd, m, k, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++)
        assert(c[i] == k);

    // matrixMul_noBankConflict
    // matrixMul_noBankConflict<<<grid, block>>>(cd, ad, bd, k, n);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(c, cd, m *n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < m *n; i++)
        assert(c[i] == k);

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    delete []a;
    delete []b;
    delete []c;
}