#include <random>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"

__global__ void bankConflictTest(float* fake_result) {
    __shared__ float sm[128];
    int i = threadIdx.x / 4;
    fake_result[i] = sm[i];
}

int main(int argc, char *argv[]) {
    float *fake_result_d;
    hipMalloc((void**)&fake_result_d, sizeof(float) * 128);
    dim3 block(128);
    dim3 grid(1);
    bankConflictTest<<<grid, block>>>(fake_result_d);
    hipFree(fake_result_d);
}